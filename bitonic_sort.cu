
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

/* Every thread gets exactly one value in the unsorted array. */
#define THREADS 512 // 2^9
#define BLOCKS 262144 // 2^15
#define NUM_VALS THREADS*BLOCKS

void print_time(clock_t start, clock_t stop)
{
  double timed = ((double) (stop - start)) / CLOCKS_PER_SEC;
  printf("Elapsed time: %.8fs\n", timed);
}

void print_array(int *array, int length) 
{
  for (int i = 0; i < length; ++i) {printf("%d ",  array[i]);}
  printf("\n");
}

void fill_array(int *array, int length)
{
  srand(time(NULL));
  for (int i = 0; i < length; ++i) {array[i] = (int)rand();}
}

__global__ void bitonic_sort_step(int *dev_values, int j, int k)
{
  unsigned int i, ixj; 
  i = threadIdx.x + blockDim.x * blockIdx.x;
  ixj = i^j;

  if ((ixj)>i) {
    if ((i&k)==0) {
      if (dev_values[i]>dev_values[ixj]) {
        int temp = dev_values[i];
        dev_values[i] = dev_values[ixj];
        dev_values[ixj] = temp;
      }
    }
    if ((i&k)!=0) {
      if (dev_values[i]<dev_values[ixj]) {
        int temp = dev_values[i];
        dev_values[i] = dev_values[ixj];
        dev_values[ixj] = temp;
      }
    }
  }
}

void bitonic_sort(int *values)
{
  int *dev_values;
  size_t size = NUM_VALS * sizeof(int);

  hipMalloc((void**) &dev_values, size);
  hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);

  dim3 blocks(BLOCKS,1);
  dim3 threads(THREADS,1);

  int j, k;
  for (k = 2; k <= NUM_VALS; k <<= 1) {
    for (j=k>>1; j>0; j=j>>1) {
      bitonic_sort_step<<<blocks, threads>>>(dev_values, j, k);
    }
  }
  hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
  hipFree(dev_values);
}

int main(void)
{
  clock_t start, stop;

  int *values = (int*) malloc( NUM_VALS * sizeof(int));
  fill_array(values, NUM_VALS);

  start = clock();
  bitonic_sort(values);
  stop = clock();
  print_array(values, NUM_VALS);

  print_time(start, stop);
}